
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<n; i += stride)
        y[i] = y[i] + x[i];
}


int main(void) {
    
    int n = 1000000;
    float *x, *y;
    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    // Initialize arrays
    for (int i=0; i<n; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(n, x, y);
    
    
    hipDeviceSynchronize();
    
    // Check for errors (all values should be 3.0f)
    int error = 0;
    for (int i=0; i<n; i++) {
        if (y[i]-3.0 != 0)
            error += 1;
    }
    
    printf("Errors: %d\n", error);

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;
}